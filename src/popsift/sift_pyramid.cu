#include "hip/hip_runtime.h"
/*
* Copyright 2016, Simula Research Laboratory
*
* This Source Code Form is subject to the terms of the Mozilla Public
* License, v. 2.0. If a copy of the MPL was not distributed with this
* file, You can obtain one at http://mozilla.org/MPL/2.0/.
*/
#include <iostream>
#include <fstream>
#include <sstream>
#include <vector>
#include <stdio.h>
#include <sys/stat.h>
#ifdef _WIN32
#include <direct.h>
#define stat _stat
#define mkdir(name, mode) _mkdir(name)
#endif

#include "sift_pyramid.h"
#include "sift_extremum.h"
#include "common/debug_macros.h"

#define PYRAMID_PRINT_DEBUG 0

using namespace std;

namespace popsift {

__global__
    void py_print_corner_float(float* img, uint32_t pitch, uint32_t height, uint32_t level)
{
    const int xbase = 0;
    const int ybase = level * height + 0;
    for (int i = 0; i<10; i++) {
        for (int j = 0; j<10; j++) {
            printf("%3.3f ", img[(ybase + i)*pitch + xbase + j]);
        }
        printf("\n");
    }
    printf("\n");
}

__global__
    void py_print_corner_float_transposed(float* img, uint32_t pitch, uint32_t height, uint32_t level)
{
    const int xbase = 0;
    const int ybase = level * height + 0;
    for (int i = 0; i<10; i++) {
        for (int j = 0; j<10; j++) {
            printf("%3.3f ", img[(ybase + j)*pitch + xbase + i]);
        }
        printf("\n");
    }
    printf("\n");
}

void Pyramid::download_and_save_array(const char* basename, uint32_t octave, uint32_t level)
{
    if (octave < _num_octaves) {
        _octaves[octave].download_and_save_array(basename, octave, level);
    }
    else {
        cerr << "Octave " << octave << " does not exist" << endl;
        return;
    }
}

void Pyramid::download_descriptors(const Config& conf, uint32_t octave)
{
    _octaves[octave].downloadDescriptor(conf);
}

void Pyramid::save_descriptors(const Config& conf, const char* basename, uint32_t octave)
{
    struct stat st = { 0 };
    if (stat("dir-desc", &st) == -1) {
        mkdir("dir-desc", 0700);
    }
    ostringstream ostr;
    ostr << "dir-desc/desc-" << basename << "-o-" << octave << ".txt";
    ofstream of(ostr.str().c_str());
    _octaves[octave].writeDescriptor(conf, of, true);

    if (stat("dir-fpt", &st) == -1) {
        mkdir("dir-fpt", 0700);
    }
    ostringstream ostr2;
    ostr2 << "dir-fpt/desc-" << basename << "-o-" << octave << ".txt";
    ofstream of2(ostr2.str().c_str());
    _octaves[octave].writeDescriptor(conf, of2, false);
}

Pyramid::Pyramid( Config& config,
                  Image* base,
                  int width,
                  int height )
    : _num_octaves( config.octaves )
    , _levels( config.levels + 3 )
    , _assume_initial_blur( config.hasInitialBlur() )
    , _initial_blur( config.getInitialBlur() )
{
    // cerr << "Entering " << __FUNCTION__ << endl;

    _octaves = new Octave[_num_octaves];

    int w = width;
    int h = height;

    // cout << "Size of the first octave's images: " << w << "X" << h << endl;

    for (int o = 0; o<_num_octaves; o++) {
        _octaves[o].debugSetOctave(o);
        _octaves[o].alloc(w, h, _levels, _gauss_group);
        w = ceilf(w / 2.0f);
        h = ceilf(h / 2.0f);
    }
}

Pyramid::~Pyramid()
{
    delete[] _octaves;
}

#define LOGTIME_0(a)
#define LOGTIME_1(a)

Features* Pyramid::find_extrema( const Config& conf,
                                 Image*        base )
{
    LOGTIME_0( hipEvent_t start );
    LOGTIME_1( hipEvent_t done_reset );
    LOGTIME_1( hipEvent_t done_pyramid );
    LOGTIME_1( hipEvent_t done_extrema );
    LOGTIME_1( hipEvent_t done_orientation );
    LOGTIME_0( hipEvent_t done_descriptors );
    LOGTIME_0( hipEvent_t done );

    LOGTIME_0( hipEventCreate( &start ) );
    LOGTIME_1( hipEventCreate( &done_reset ) );
    LOGTIME_1( hipEventCreate( &done_pyramid ) );
    LOGTIME_1( hipEventCreate( &done_extrema ) );
    LOGTIME_1( hipEventCreate( &done_orientation ) );
    LOGTIME_0( hipEventCreate( &done_descriptors ) );
    LOGTIME_0( hipEventCreate( &done ) );

    LOGTIME_0( hipDeviceSynchronize() );
    LOGTIME_0( hipEventRecord( start ) );

    reset_extrema_mgmt( );

    LOGTIME_1( hipDeviceSynchronize() );
    LOGTIME_1( hipEventRecord( done_reset ) );

    build_pyramid( conf, base );

    LOGTIME_1( hipDeviceSynchronize() );
    LOGTIME_1( hipEventRecord( done_pyramid ) );

    find_extrema( conf );

    LOGTIME_1( hipDeviceSynchronize() );
    LOGTIME_1( hipEventRecord( done_extrema ) );

    orientation( conf );

    LOGTIME_1( hipDeviceSynchronize() );
    LOGTIME_1( hipEventRecord( done_orientation ) );

    descriptors( conf );

    LOGTIME_0( hipDeviceSynchronize() );
    LOGTIME_0( hipEventRecord( done_descriptors ) );

    Features* features        = new Features;
    int       num_extrema     = 0;
    int       num_descriptors = 0;
    for (int o = 0; o<_num_octaves; o++) {
        // synchronous download of number of extrema and number of descriptors
        _octaves[o].readExtremaCount();

        // asynchronous download of extrema and descriptors (in stream 0)
        _octaves[o].downloadDescriptor(conf);

        num_extrema += _octaves[o].getExtremaCount();
        num_descriptors += _octaves[o].getDescriptorCount();
    }

    LOGTIME_0( hipDeviceSynchronize() );
    LOGTIME_0( hipEventRecord( done ) );

    LOGTIME_0( hipDeviceSynchronize() );
    LOGTIME_1( float start_reset = 0 );
    LOGTIME_1( float start_pyramid = 0 );
    LOGTIME_1( float start_extrema = 0 );
    LOGTIME_1( float start_orientation = 0 );
    LOGTIME_0( float start_descriptors = 0 );
    LOGTIME_0( float start_done = 0 );

    LOGTIME_1( hipEventElapsedTime( &start_reset, start, done_reset ) );
    LOGTIME_1( hipEventElapsedTime( &start_pyramid, start, done_pyramid ) );
    LOGTIME_1( hipEventElapsedTime( &start_extrema, start, done_extrema ) );
    LOGTIME_1( hipEventElapsedTime( &start_orientation, start, done_orientation ) );
    LOGTIME_0( hipEventElapsedTime( &start_descriptors, start, done_descriptors ) );
    LOGTIME_0( hipEventElapsedTime( &start_done, start, done ) );

    LOGTIME_0( cerr << "Time passed from start to" << endl );
    LOGTIME_1( cerr << " - reset:       " << start_reset << " ms" << endl );
    LOGTIME_1( cerr << " - pyramid:     " << start_pyramid << " ms" << endl );
    LOGTIME_1( cerr << " - extrema:     " << start_extrema << " ms" << endl );
    LOGTIME_1( cerr << " - orientation: " << start_orientation << " ms" << endl );
    LOGTIME_0( cerr << " - descriptors: " << start_descriptors << " ms" << endl );
    LOGTIME_0( cerr << " - downloaded:  " << start_done << " ms" << endl );

    LOGTIME_0( hipEventDestroy( start ) );
    LOGTIME_1( hipEventDestroy( done_reset ) );
    LOGTIME_1( hipEventDestroy( done_pyramid ) );
    LOGTIME_1( hipEventDestroy( done_extrema ) );
    LOGTIME_1( hipEventDestroy( done_orientation ) );
    LOGTIME_0( hipEventDestroy( done_descriptors ) );
    LOGTIME_0( hipEventDestroy( done ) );

    features->_features.resize( num_extrema );

    features->_num_descriptors = num_descriptors;
    features->_desc_buffer = new Descriptor[num_descriptors];

    // ensure that asynchronous downloads are finished
    hipDeviceSynchronize();

    num_extrema = 0;
    num_descriptors = 0;
    for (int o = 0; o<_num_octaves; o++) {
        if (num_extrema < features->_features.size()) {
            Feature*    feature_base = &features->_features[num_extrema];
            Descriptor* desc_base = &features->_desc_buffer[num_descriptors];
            _octaves[o].copyExtrema(conf, feature_base, desc_base);
        }
        else {
            assert(_octaves[o].getExtremaCount() == 0);
        }

        num_extrema += _octaves[o].getExtremaCount();
        num_descriptors += _octaves[o].getDescriptorCount();
    }

    return features;
}

void Pyramid::reset_extrema_mgmt()
{
    for (int o = 0; o<_num_octaves; o++) {
        _octaves[o].reset_extrema_mgmt();
    }
}

} // namespace popsift
