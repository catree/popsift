#include "hip/hip_runtime.h"
/*
 * Copyright 2016-2017, Simula Research Laboratory
 *
 * This Source Code Form is subject to the terms of the Mozilla Public
 * License, v. 2.0. If a copy of the MPL was not distributed with this
 * file, You can obtain one at http://mozilla.org/MPL/2.0/.
 */
#include <stdio.h>
#include <iso646.h>

#include "sift_constants.h"
#include "s_gradiant.h"
#include "s_desc_notile.h"
#include "common/assist.h"
#include "common/vec_macros.h"

using namespace popsift;

__device__ static inline
void ext_desc_get_grad( const float                  x,
                        const float                  y,
                        const int                    level,
                        hipTextureObject_t          texLinear,
                        const float                  cos_t,
                        const float                  sin_t,
                        const float                  SBP,
                        const int                    offx,
                        const int                    offy,
                        float&                       mod,
                        float&                       th )
{
    const float mvx = -2.5f + offx/8.0f + 1.0f/16.0f;
    const float mvy = -2.5f + offy/8.0f + 1.0f/16.0f;
    const float ptx  = ( cos_t * mvx - sin_t * mvy ) * SBP;
    const float pty  = ( cos_t * mvy + sin_t * mvx ) * SBP;
    get_gradiant( mod, th, x + ptx, y + pty, cos_t, sin_t, texLinear, level );
    th += ( th <  0.0f  ? M_PI2 : 0.0f ); //  if (th <  0.0f ) th += M_PI2;
    th -= ( th >= M_PI2 ? M_PI2 : 0.0f ); //  if (th >= M_PI2) th -= M_PI2;
}

__device__ static inline
void ext_desc_inc_tile( float* dpt, const int ix, const int iy, const int xd, const int yd, const float th, const float mod, const float ww )
{
    if( ix < 0 || iy < 0 || ix > 3 || iy > 3 ) return;

    const float wx = d_consts.desc_tile[xd];
    const float wy = d_consts.desc_tile[yd];

    const float  wgt = ww * wx * wy * mod;

    const float tth  = th * M_4RPI;
    const int   fo   = (int)floorf(tth);
    const float do0  = tth - fo;
    const float wgt1 = 1.0f - do0;
    const float wgt2 = do0;

    const int fo0  =   fo       & (8-1); // % 8;
    const int fo1  = ( fo + 1 ) & (8-1); // % 8;
    dpt[fo0] += ( wgt * wgt1 );
    dpt[fo1] += ( wgt * wgt2 );
}

__device__ static inline
void ext_desc_notile_sub( const float                  x,
                          const float                  y,
                          const int                    level,
                          const float                  cos_t,
                          const float                  sin_t,
                          const float                  SBP,
                          const Extremum* __restrict__ ext,
                          float* __restrict__          features,
                          hipTextureObject_t          texLinear )
{
    float dpt[2][32];

    {
        int iy = 0;
        memset( dpt[0], 0, 32*sizeof(float) );
        for( int ix=0; ix<5; ix++ ) {
            for( int yd = threadIdx.x / 8; yd < 8; yd += 4 ) {
                const int xd = threadIdx.x & (8-1);
                const int offx = ix*8+xd;
                const int offy = iy*8+yd;
                float mod, th;
                ext_desc_get_grad( x, y, level, texLinear, cos_t, sin_t, SBP, offx, offy, mod, th );
                float ww = d_consts.desc_gauss[offy][offx];
                int tile;

                tile = ix-1;
                ext_desc_inc_tile( &dpt[0][tile*8], ix-1, iy,   xd+8, yd,   th, mod, ww );

                tile = ix;
                ext_desc_inc_tile( &dpt[0][tile*8], ix,   iy,   xd,   yd,   th, mod, ww );
            }
        }
    }
    for( int iy=1; iy<5; iy++ ) {
        memset( dpt[iy&1?1:0], 0, 32*sizeof(float) );
        for( int ix=0; ix<5; ix++ ) {
            for( int yd = threadIdx.x / 8; yd < 8; yd += 4 ) {
                const int xd = threadIdx.x & (8-1);
                const int offx = ix*8+xd;
                const int offy = iy*8+yd;
                float mod, th;
                ext_desc_get_grad( x, y, level, texLinear, cos_t, sin_t, SBP, offx, offy, mod, th );
                float ww = d_consts.desc_gauss[offy][offx];
                int tile;

                tile = ix-1;
                ext_desc_inc_tile( &dpt[iy&1?0:1][tile*8], ix-1, iy-1, xd+8, yd+8, th, mod, ww );

                tile = ix;
                ext_desc_inc_tile( &dpt[iy&1?0:1][tile*8], ix,   iy-1, xd,   yd+8, th, mod, ww );

                tile = ix-1;
                ext_desc_inc_tile( &dpt[iy&1?1:0][tile*8], ix-1, iy,   xd+8, yd,   th, mod, ww );

                tile = ix;
                ext_desc_inc_tile( &dpt[iy&1?1:0][tile*8], ix,   iy,   xd,   yd,   th, mod, ww );
            }
        }

        __syncthreads();

        for( int i=0; i<32; i++ ) {
            float d = dpt[iy&1?0:1][i];
            d += __shfl_xor( d,  1 );
            d += __shfl_xor( d,  2 );
            d += __shfl_xor( d,  4 );
            d += __shfl_xor( d,  8 );
            d += __shfl_xor( d, 16 );
            dpt[iy&1?0:1][i] = d;
        }

        __syncthreads();

        features[(iy-1)*32+threadIdx.x] = dpt[iy&1?0:1][threadIdx.x];
    }
}

__global__
void ext_desc_notile( Extremum*           extrema,
                      const int           num,
                      Descriptor*         descs,
                      int*                feat_to_ext_map,
                      hipTextureObject_t texLinear )
{
    const int   offset   = blockIdx.x * blockDim.z + threadIdx.z;
    if( offset >= num ) return;

    Descriptor* desc     = &descs[offset];
    const int   ext_idx  = feat_to_ext_map[offset];
    Extremum*   ext      = &extrema[ext_idx];
    const int   ext_base = ext->idx_ori;
    const int   ext_num  = offset - ext_base;
    const float ang      = ext->orientation[ext_num];

    if( ext->sigma == 0 ) return;
    const float SBP   = fabsf(DESC_MAGNIFY * ext->sigma);

    float cos_t;
    float sin_t;
    __sincosf( ang, &sin_t, &cos_t );

    ext_desc_notile_sub( ext->xpos, ext->ypos, ext->lpos,
                         cos_t, sin_t, SBP,
                         ext,
                         desc->features,
                         texLinear );
}

