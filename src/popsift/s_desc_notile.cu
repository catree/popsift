#include "hip/hip_runtime.h"
/*
 * Copyright 2016-2017, Simula Research Laboratory
 *
 * This Source Code Form is subject to the terms of the Mozilla Public
 * License, v. 2.0. If a copy of the MPL was not distributed with this
 * file, You can obtain one at http://mozilla.org/MPL/2.0/.
 */
#include <stdio.h>
#include <iso646.h>

#include "sift_constants.h"
#include "s_gradiant.h"
#include "s_desc_notile.h"
#include "common/assist.h"
#include "common/vec_macros.h"

using namespace popsift;

__device__ static inline
void ext_desc_notile_sub( const float                  x,
                          const float                  y,
                          const int                    level,
                          const float                  cos_t,
                          const float                  sin_t,
                          const float                  SBP,
                          const Extremum* __restrict__ ext,
                          float* __restrict__          features,
                          hipTextureObject_t          texLinear )
{
    const int ix   = threadIdx.y;
    const int iy   = threadIdx.z;
    const int tile = ( iy << 2 ) + ix;

    __shared__ float dpt[128];
    if( threadIdx.z < 2 ) {
        dpt[threadIdx.z * 64 + threadIdx.y * 16 + threadIdx.x] = 0.0f;
    }
    __syncthreads();

    const float csbp  = cos_t * SBP;
    const float ssbp  = sin_t * SBP;

    const float2 offset = make_float2( ix - 1.5f, iy - 1.5f );

    const float2 pt = make_float2( ::fmaf( csbp, offset.x, ::fmaf( -ssbp, offset.y, x ) ),
                                   ::fmaf( csbp, offset.y, ::fmaf(  ssbp, offset.x, y ) ) );

    // float dpt[8] = { 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f, 0.0f };

    const float2 lft_dn = make_float2( -cos_t + sin_t, -cos_t - sin_t );
    const float2 rgt_stp = make_float2(  cos_t, sin_t ) / 8.0f;
    const float2 up__stp = make_float2( -sin_t, cos_t ) / 8.0f;

    int xd = threadIdx.x;
    for( int yd=0; yd<16; yd++ )
    {
        float2 pixo = lft_dn + (xd+0.5f) * rgt_stp + (yd+0.5f) * up__stp;
        float2 pix  = pixo * SBP;

        float mod;
        float th;
        get_gradiant( mod, th, (pt+pix).x, (pt+pix).y, cos_t, sin_t, texLinear, level );
        th += ( th <  0.0f  ? M_PI2 : 0.0f ); //  if (th <  0.0f ) th += M_PI2;
        th -= ( th >= M_PI2 ? M_PI2 : 0.0f ); //  if (th >= M_PI2) th -= M_PI2;

        const float ww = d_consts.desc_gauss[iy*8+yd][ix*8+xd];
        const float wx = d_consts.desc_tile[xd];
        const float wy = d_consts.desc_tile[yd];

        const float  wgt = ww * wx * wy * mod;

        const float tth  = __fmul_ru( th, M_4RPI ); // th * M_4RPI;
        const int   fo   = (int)floorf(tth);
        const float do0  = tth - fo;
        const float wgt1 = 1.0f - do0;
        const float wgt2 = do0;

        int fo0  =   fo       % 8;
        int fo1  = ( fo + 1 ) % 8;
        atomicAdd( &dpt[tile*8+fo0], wgt * wgt1 );
        atomicAdd( &dpt[tile*8+fo1], wgt * wgt2 );
        __syncthreads();
    }

    if( threadIdx.z < 2 ) {
        const int idx = threadIdx.z * 64 + threadIdx.y * 16 + threadIdx.x;
        features[idx] = dpt[idx];
    }
}

__global__
void ext_desc_notile( Extremum*           extrema,
                      Descriptor*         descs,
                      int*                feat_to_ext_map,
                      hipTextureObject_t texLinear )
{
    const int   offset   = blockIdx.x;
    Descriptor* desc     = &descs[offset];
    const int   ext_idx  = feat_to_ext_map[offset];
    Extremum*   ext      = &extrema[ext_idx];
    const int   ext_base = ext->idx_ori;
    const int   ext_num  = offset - ext_base;
    const float ang      = ext->orientation[ext_num];

    if( ext->sigma == 0 ) return;
    const float SBP   = fabsf(DESC_MAGNIFY * ext->sigma);

    float cos_t;
    float sin_t;
    __sincosf( ang, &sin_t, &cos_t );

    ext_desc_notile_sub( ext->xpos, ext->ypos, ext->lpos,
                         cos_t, sin_t, SBP,
                         ext,
                         desc->features,
                         texLinear );
}

